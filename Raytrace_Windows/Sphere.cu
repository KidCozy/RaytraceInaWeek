#include "hip/hip_runtime.h"
#include "Sphere.cuh"


Sphere::Sphere()
{
}


Sphere::~Sphere()
{
}

bool Sphere::hit(const Ray & r, float t_min, float t_max, HitRecord & rec) const
{
	Vector3 oc = r.Origin() - Center;
	float a = dot(r.Direction(), r.Direction());
	float b = dot(oc, r.Direction());
	float c = dot(oc, oc) - Radius * Radius;
	float discriminant = b * b - a * c;

	//std::cout << a << std::endl;

	if (discriminant > 0)
	{
		float temp = (-b - sqrt(discriminant)) / a;
		if (temp < t_max && temp > t_min)
		{
			rec.t = temp;
			rec.p = r.PointAtParameter(rec.t);
			rec.normal = (rec.p - Center) / Radius;
			rec.matptr = matptr;
			return true;
		}
		temp = (-b + sqrt(discriminant)) / a;
		if (temp < t_max && temp > t_min)
		{
			rec.t = temp;
			rec.p = r.PointAtParameter(rec.t);
			rec.normal = (rec.p - Center) / Radius;
			rec.matptr = matptr;
			return true;
		}

	}

	return false;
}
