#include "hip/hip_runtime.h"
#include"CUDARender.cuh"

#include"hip/hip_runtime.h"
#include<iostream>
#include<hipfft/hipfft.h>
#include"hipblas.h"
#include<stdio.h>
#include<stdlib.h>


bool CUDARender::Init(HDC dc)
{
	MainDC = GetDC(hwnd);
	MemoryDC = CreateCompatibleDC(MainDC);
	
	BITMAPINFO ScreenDesc{};
	ScreenDesc.bmiHeader.biSize = sizeof(BITMAPINFOHEADER);
	ScreenDesc.bmiHeader.biWidth = WIDTH;
	ScreenDesc.bmiHeader.biHeight = -HEIGHT;
	ScreenDesc.bmiHeader.biPlanes = 1;
	ScreenDesc.bmiHeader.biBitCount = BITS_PER_PIXEL;
	ScreenDesc.bmiHeader.biCompression = BI_RGB;

	DIBitmap = CreateDIBSection(MemoryDC, &ScreenDesc, DIB_RGB_COLORS, (LPVOID*)&ScreenBits, NULL, NULL);
	OldDIBitmap = (HBITMAP)SelectObject(MemoryDC, DIBitmap);


	Timer = time(NULL);

	return true;
}

bool CUDARender::Update()
{
//	Clear(VColor(0.0f,0.0f,0.0f));

	if (RenderDone)
		return false;

	Vector3 LowLeftCorner(-2.0f, -1.5f, -1.0f);
	Hittable* List[5];
	Camera Cam(Vector3(-2, 2, 1), Vector3(0, 0, -1), Vector3(0, 1, 0), 60, ASPECTRATIO);
	float R = cos(M_PI / 4);

	List[0] = new Sphere(Vector3(0, 0, -1), 0.5f, new Lambertian(Vector3(0.1f, 0.2f, 0.5f)));
	List[1] = new Sphere(Vector3(0, -100.5f, -1), 100.0f, new Lambertian(Vector3(0.8f, 0.8f, 0.0f)));
	List[2] = new Sphere(Vector3(1, 0, -1), 0.5f, new Metal(Vector3(0.8f, 0.6f, 0.2f), 0.3f));
	List[3] = new Sphere(Vector3(-1, 0, -1), 0.5f, new Dielectric(1.5f));
	List[4] = new Sphere(Vector3(-1, 0, -1), -0.45f, new Dielectric(1.5f));
	Hittable* World = new HittableList(List, 5);


	// ������ ���� ������ ������ ���ϴ� ����Ʈ ������ �����Ѵ�.
	// ���� ��� (800, 600) �̹Ƿ� 4:3 ������ ���Ͽ���.
	Vector3 Horizontal(4.0f, 0.0f, 0.0f);
	Vector3 Vertical(0.0f, 3.0f, 0.0f);
	Vector3 Origin(0.0f, 0.0f, 0.0f);

	//OutImage << "P3\n" << WIDTH << " " << HEIGHT << "\n255\n";

	for (int y = HEIGHT - 1; y >= 0; y--)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			Vector3 color(0.0f, 0.0f, 0.0f);

			for (int s = 0; s < SAMPLE; s++)
			{
				float u = float(x + random_double()) / float(WIDTH);
				float v = float(y + random_double()) / float(HEIGHT);

				Ray ray = Cam.GetRay(u, v);

				color += Color(ray, World, 0);
			}

			color /= float(SAMPLE);
			color = Vector3(sqrt(color[0]), sqrt(color[1]), sqrt(color[2]));

			int ir = int(255.99*color[0]);
			int ig = int(255.99*color[1]);
			int ib = int(255.99*color[2]);

			ScreenColors[y][x].SetRGB(ir, ig, ib);
		}
	}

	ElapsedTime = (double)(time(NULL) - Timer);

	//OutImage.close();

	return true;
}

bool CUDARender::Render()
{
	SwapBuffer();
	if (RenderDone)
		return false;


	for (int y = 0; y < HEIGHT; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{

			
			CurrentColor.SetRGB(ScreenColors[HEIGHT - y][x]);

			SetPixel(x, y);

		}
	}

	printf("%f", ElapsedTime);

	

	RenderDone = true;

	return true;
}

bool CUDARender::Release()
{
	SelectObject(MemoryDC, OldDIBitmap);
	DeleteObject(DIBitmap);
	ReleaseDC(hwnd, MemoryDC);
	return true;
}

Vector3 CUDARender::Color(const Ray & ray, Hittable * World, int depth)
{
	HitRecord rec;

	if (World->hit(ray, 0.001f, FLT_MAX, rec))
	{
		Ray scattered;
		Vector3 attenuation;

		if (depth < 50 && rec.matptr->Scatter(ray, rec, attenuation, scattered))
			return attenuation * Color(scattered, World, depth + 1);
		else
			return Vector3(0, 0, 0);
		Vector3 Target = rec.p + rec.normal + RandomInUnitSphere();
		return 0.5f* Color(Ray(rec.p, Target - rec.p), World, 0);
	}
	else
	{
		Vector3 Direction = unit_vector(ray.Direction());
		float t = 0.5f * (Direction.y() + 1.0f);
		return (1.0f - t) * Vector3(1.0f, 1.0f, 1.0f) + t * Vector3(0.5f, 0.7f, 1.0f);
	}
}

float CUDARender::HitSphere(const Vector3 & center, float radius, const Ray & ray)
{
	Vector3 oc = ray.Origin() - center;

	float a = dot(ray.Direction(), ray.Direction());
	float b = 2.0f * dot(oc, ray.Direction());
	float c = dot(oc, oc) - radius * radius;

	float discriminant = b * b - 4 * a*c;

	if (discriminant < 0)
		return -1.0;

	else
		return (-b - sqrt(discriminant)) / (2.0*a);
}

void CUDARender::Clear(VColor ClearColor)
{
	

	UINT Offset = 0;
	while (Offset < BytesPerScanline)
	{
		*((DWORD*)(ScreenBits + Offset)) = *(DWORD*)ClearColor.GetRGB(); // 0x00000000
		Offset += BYTES_PER_PIXEL;

	}

	Offset = BytesPerScanline;


	for (int i = 0; i < HEIGHT - 1; i++)
	{
		memcpy(ScreenBits + Offset, ScreenBits, BytesPerScanline);
		Offset += BytesPerScanline;
	}

}

void CUDARender::SwapBuffer()
{
	BitBlt(MainDC, 0, 0, WIDTH, HEIGHT, MemoryDC, 0, 0, SRCCOPY);
}

void CUDARender::SetPixel(UINT x, UINT y)
{

	if (!IsInScreen(x, y))
		return;

	int Offset = (y * BytesPerScanline) + (x * BYTES_PER_PIXEL);

	*(ScreenBits + Offset + 0) = CurrentColor._0;
	*(ScreenBits + Offset + 1) = CurrentColor._1;
	*(ScreenBits + Offset + 2) = CurrentColor._2;
}

bool CUDARender::IsInScreen(UINT x, UINT y)
{
	if (x > WIDTH || x < 0 || y > HEIGHT || y < 0)
		return false;

	return true;
		
}

CUDARender::CUDARender()
{
}

CUDARender::~CUDARender()
{
}
