#include "hip/hip_runtime.h"
#include"CUDARender.cuh"

#include"hip/hip_runtime.h"
#include<iostream>
#include<hipfft/hipfft.h>
#include"hipblas.h"
#include<stdio.h>
#include<stdlib.h>


bool CUDARender::Init(HDC dc)
{
	MainDC = dc;
	//OutImage = std::ofstream("image.png");
	//AllocConsole();

	Timer = time(NULL);

	return true;
}

bool CUDARender::Update()
{
	if (RenderDone)
		return false;

	Vector3 LowLeftCorner(-2.0f, -1.5f, -1.0f);
	Hittable* List[5];
	Camera Cam(Vector3(-2, 2, 1), Vector3(0, 0, -1), Vector3(0, 1, 0), 60, ASPECTRATIO);
	float R = cos(M_PI / 4);

	List[0] = new Sphere(Vector3(0, 0, -1), 0.5f, new Lambertian(Vector3(0.1f, 0.2f, 0.5f)));
	List[1] = new Sphere(Vector3(0, -100.5f, -1), 100.0f, new Lambertian(Vector3(0.8f, 0.8f, 0.0f)));
	List[2] = new Sphere(Vector3(1, 0, -1), 0.5f, new Metal(Vector3(0.8f, 0.6f, 0.2f), 0.3f));
	List[3] = new Sphere(Vector3(-1, 0, -1), 0.5f, new Dielectric(1.5f));
	List[4] = new Sphere(Vector3(-1, 0, -1), -0.45f, new Dielectric(1.5f));
	Hittable* World = new HittableList(List, 5);


	// ������ ���� ������ ������ ���ϴ� ����Ʈ ������ �����Ѵ�.
	// ���� ��� (800, 600) �̹Ƿ� 4:3 ������ ���Ͽ���.
	Vector3 Horizontal(4.0f, 0.0f, 0.0f);
	Vector3 Vertical(0.0f, 3.0f, 0.0f);
	Vector3 Origin(0.0f, 0.0f, 0.0f);

	//OutImage << "P3\n" << WIDTH << " " << HEIGHT << "\n255\n";

	for (int y = HEIGHT - 1; y >= 0; y--)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			Vector3 color(0.0f, 0.0f, 0.0f);

			for (int s = 0; s < SAMPLE; s++)
			{
				float u = float(x + random_double()) / float(WIDTH);
				float v = float(y + random_double()) / float(HEIGHT);

				Ray ray = Cam.GetRay(u, v);

				color += Color(ray, World, 0);
			}

			color /= float(SAMPLE);
			color = Vector3(sqrt(color[0]), sqrt(color[1]), sqrt(color[2]));

			int ir = int(255.99*color[0]);
			int ig = int(255.99*color[1]);
			int ib = int(255.99*color[2]);

			//	OutImage << ir << " " << ig << " " << ib << "\n";

			ScreenColors[y][x] = RGB(ir, ig, ib);
		}
	}

	ElapsedTime = (double)(time(NULL) - Timer);

	//OutImage.close();

	return true;
}

bool CUDARender::Render()
{

	if (RenderDone)
		return false;


	for (int y = 0; y < HEIGHT; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{

			SetPixel(MainDC, x, y, ScreenColors[HEIGHT - y][x]);

		}
	}

	//wchar_t Buffer[256];
	//wsprintf(Buffer, L"%f", ElapsedTime);
	//TextOutW(MainDC, 0, 0, Buffer, 1);
	//std::cout << ElapsedTime << std::endl;
	printf("%f", ElapsedTime);

	RenderDone = true;

	return true;
}

bool CUDARender::Release()
{
	return true;
}

Vector3 CUDARender::Color(const Ray & ray, Hittable * World, int depth)
{
	HitRecord rec;

	if (World->hit(ray, 0.001f, FLT_MAX, rec))
	{
		Ray scattered;
		Vector3 attenuation;

		if (depth < 50 && rec.matptr->Scatter(ray, rec, attenuation, scattered))
			return attenuation * Color(scattered, World, depth + 1);
		else
			return Vector3(0, 0, 0);
		Vector3 Target = rec.p + rec.normal + RandomInUnitSphere();
		return 0.5f* Color(Ray(rec.p, Target - rec.p), World, 0);
	}
	else
	{
		Vector3 Direction = unit_vector(ray.Direction());
		float t = 0.5f * (Direction.y() + 1.0f);
		return (1.0f - t) * Vector3(1.0f, 1.0f, 1.0f) + t * Vector3(0.5f, 0.7f, 1.0f);
	}
}

float CUDARender::HitSphere(const Vector3 & center, float radius, const Ray & ray)
{
	Vector3 oc = ray.Origin() - center;

	float a = dot(ray.Direction(), ray.Direction());
	float b = 2.0f * dot(oc, ray.Direction());
	float c = dot(oc, oc) - radius * radius;

	float discriminant = b * b - 4 * a*c;

	if (discriminant < 0)
		return -1.0;

	else
		return (-b - sqrt(discriminant)) / (2.0*a);
}

CUDARender::CUDARender()
{
}

CUDARender::~CUDARender()
{
}
