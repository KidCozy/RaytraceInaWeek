#include "hip/hip_runtime.h"
#pragma once
#include"Material.cuh"


Vector3 RandomInUnitSphere()
{
	Vector3 p;

	do
	{
		p = 2.0f * Vector3(random_double(), random_double(), random_double()) - Vector3(1.0f, 1.0f, 1.0f);
	} while (p.squared_length() >= 1.0f);

	return p;
}

Vector3 Reflect(const Vector3& v, const Vector3& n)
{
	return v - 2 * dot(v, n)*n;
}

bool Refract(const Vector3 & v, const Vector3& n, float niovernt, Vector3 & refracted)
{
	Vector3 uv = unit_vector(v);
	float dt = dot(uv, n);
	float discriminant = 1.0f - niovernt * niovernt * (1 - dt * dt);

	if (discriminant > 0)
	{
		refracted = niovernt * (uv - n * dt) - n * sqrt(discriminant);
		return true;
	}
	else
		return false;
}

float Schlick(float cosine, float ref_idx)
{
	float r0 = (1 - ref_idx) / (1 + ref_idx);
	r0 = r0 * r0;

	return r0 + (1 - r0)*pow((1 - cosine), 5);
}

bool Lambertian::Scatter(const Ray & rayin, const HitRecord & rec, Vector3 & attenuation, Ray & scattered) const
{
	Vector3 Target = rec.p + rec.normal + RandomInUnitSphere();
	scattered = Ray(rec.p, Target - rec.p);
	attenuation = Albedo;
	return true;
}

bool Metal::Scatter(const Ray & rayin, const HitRecord & rec, Vector3 & attenuation, Ray & scattered) const
{
	Vector3 Reflected = Reflect(unit_vector(rayin.Direction()), rec.normal);
	scattered = Ray(rec.p, Reflected);
	attenuation = Albedo;

	return (dot(scattered.Direction(), rec.normal) > 0);
}

bool Dielectric::Scatter(const Ray & rayin, const HitRecord & rec, Vector3 & attenuation, Ray & scattered) const
{
	float NioverNt;
	Vector3 OutwardNormal;
	Vector3 Reflected = Reflect(rayin.Direction(), rec.normal);
	Vector3 Refracted;

	float ReflectProb;
	float Cosine;

	attenuation = Vector3(1.0f, 1.0f, 1.0f);

	if (dot(rayin.Direction(), rec.normal) > 0)
	{
		OutwardNormal = -rec.normal;
		NioverNt = RefIdx;
		Cosine = RefIdx * dot(rayin.Direction(), rec.normal) / rayin.Direction().legnth();
		Cosine = sqrt(1 - RefIdx * RefIdx*(1 - Cosine * Cosine));
	}
	else
	{
		OutwardNormal = rec.normal;
		NioverNt = 1.0f / RefIdx;

		Cosine = -dot(rayin.Direction(), rec.normal) / rayin.Direction().legnth();

	}

	if (Refract(rayin.Direction(), OutwardNormal, NioverNt, Refracted))
	{
		ReflectProb = Schlick(Cosine, RefIdx);
	}
	else
	{
		ReflectProb = 1.0f;
	}

	if (random_double() < ReflectProb)
	{
		scattered = Ray(rec.p, Reflected);
	}
	else
	{
		scattered = Ray(rec.p, Refracted);
	}

	return true;
}
